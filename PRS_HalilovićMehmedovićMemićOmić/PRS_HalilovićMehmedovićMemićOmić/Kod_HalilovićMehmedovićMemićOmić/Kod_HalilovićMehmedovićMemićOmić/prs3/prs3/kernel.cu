#include "hip/hip_runtime.h"
﻿#include<hip/hip_complex.h>
#include<iostream>
#include<math.h>
#include<hip/hip_math_constants.h>
#include<stdio.h>
#include <time.h>
#include ""
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>

using namespace std;


__host__ __device__ hipDoubleComplex complexp(double exp) {
    double a = cos(exp);
    double bi = sin(exp);
    return make_hipDoubleComplex(a, bi);
}

__global__ void fft(hipDoubleComplex* A, long int m) {
    unsigned int th = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = th / (m / 2);
    unsigned int j = th % (m / 2);
    hipDoubleComplex w = complexp(((2 * HIP_PI) / m) * j);
    hipDoubleComplex t = hipCmul(w, A[k + j + m / 2]);
    hipDoubleComplex u = A[k + j];
    A[k + j] = hipCadd(u, t);
    A[k + j + m / 2] = hipCsub(u, t);
}


__global__ void bit_reverse_copy(hipDoubleComplex* A, long int size, hipDoubleComplex* R) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n > size) return;
    int s = (int)log2((double)size);
    int revn = 0;
    for (int i = 0; i < s; i++) {
        revn += ((n >> i) & 1) << ((s - 1) - i);
    }
    hipDoubleComplex aux = A[n];
    R[revn] = aux;
}

int main() {
    int p;

   

    p = 19;
        long int n = (long int)pow(2, p);
        size_t size = n * sizeof(hipDoubleComplex);
        hipDoubleComplex* A = (hipDoubleComplex*)malloc(size);


        for (long int k = 0; k < n; k++) {
            if (k < n / 2) {
                A[k].x = 1;
                A[k].y = 0;
            }
            else {
                A[k].x = 0;
                A[k].y = 0;
            }
        }

        hipEvent_t stt, stp, stt2, stp2;
        hipEventCreate(&stt);
        hipEventCreate(&stp);
        hipEventCreate(&stt2);
        hipEventCreate(&stp2);

        hipEventRecord(stt);


        hipDoubleComplex* A_d, * B_d;
        hipMalloc(&A_d, size);
        hipMalloc(&B_d, size);
        hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);


        hipEventRecord(stt2);


        unsigned int t = (n) > 512 ? 512 : (n);
        unsigned int bt = (unsigned int)((n) / t);
        dim3 g(t);
        dim3 b(bt);


        bit_reverse_copy << <g, b >> > (A_d, n, B_d);

        long int m = 2;
        for (int i = 1; i <= log2((double)n); i++) {
            unsigned int x = ((n / 2) < 512) ? (n / 2) : 512;
            unsigned int bx = ((n / 2) / x);
            dim3 grid(x);
            dim3 blocks(bx);
            fft << <grid, blocks >> > (B_d, m);
            m *= 2;
        }
        hipEventRecord(stp2);
        hipMemcpy(A, B_d, size, hipMemcpyDeviceToHost);

        hipFree(A_d);
        hipFree(B_d);

        hipEventRecord(stp);

        hipEventSynchronize(stp);
        hipEventSynchronize(stp2);

        float milliseconds = 0;
        float milliseconds2 = 0;
        hipEventElapsedTime(&milliseconds, stt, stp);
        hipEventElapsedTime(&milliseconds2, stt2, stp2);


        cout << milliseconds << " seconds elapsed! (With copy)" << endl;
        cout << milliseconds2 << " seconds elapsed! (Without copy)" << endl;


        printf("Effective Bandwidth (GB/s): %f\n", ((n * sizeof(hipDoubleComplex) * 2) / milliseconds) / 1e6);

        printf("Computational Throughput (GB/s): %f\n", ((n * 2) / milliseconds) / 1e6);


        FILE* out;
        out = fopen("teste_cuda.txt", "a+");
        if (out) {
            fprintf(out, "%lf  -  %lf - %f - %f\n", milliseconds, milliseconds2, ((n * 4 * 3) / milliseconds) / 1e6, ((n * 2) / milliseconds) / 1e6);
            fclose(out);
        free(A);

        cout << "End" << endl;
        //int d;
        //cin >> d;
    }
}